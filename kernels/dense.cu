
#include <hip/hip_runtime.h>
extern "C" __global__ void matmul(float *A, float *B, float *C, int widthA, int widthB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < widthA && col < widthB) {
        float sum = 0;
        for (int k = 0; k < widthA; ++k) {
            sum += A[row * widthA + k] * B[k * widthB + col];
        }
        C[row * widthB + col] = sum;
    }
}